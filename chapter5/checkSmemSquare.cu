#include "hip/hip_runtime.h"
#include "../common.h"
#include <../include/hip/hip_runtime.h>
#include <stdio.h>

/*
 * An example of using shared memory to transpose square thread coordinates
 * of a CUDA grid into a global memory array. Different kernels below
 * demonstrate performing reads and writes with different ordering, as well as
 * optimizing using memory padding.
 */

#define BDIMX 16
#define BDIMY 16

void printData(char *msg, int *in,  const int size)
{
    printf("%s: ", msg);

    for (int i = 0; i < size; i++)
    {
        printf("%5d", in[i]);
        fflush(stdout);
    }

    printf("\n");
    return;
}

__global__ void setRowReadRow (int *out)
{
    // static shared memory
    __shared__ int tile[BDIMY][BDIMX];

    // mapping from thread index to global memory index
    unsigned int idx = threadIdx.y * blockDim.x + threadIdx.x;

    // shared memory store operation
    tile[threadIdx.y][threadIdx.x] = idx;

    // wait for all threads to complete
    __syncthreads();

    // shared memory load operation
    out[idx] = tile[threadIdx.y][threadIdx.x] ;
}

__global__ void setColReadCol (int *out)
{
    // static shared memory
    __shared__ int tile[BDIMX][BDIMY];

    // mapping from thread index to global memory index
    unsigned int idx = threadIdx.y * blockDim.x + threadIdx.x;

    // shared memory store operation
    tile[threadIdx.x][threadIdx.y] = idx;

    // wait for all threads to complete
    __syncthreads();

    // shared memory load operation
    out[idx] = tile[threadIdx.x][threadIdx.y];
}

__global__ void setColReadRow(int *out)
{
    // static shared memory
    __shared__ int tile[BDIMX][BDIMY];

    // mapping from thread index to global memory index
    unsigned int idx = threadIdx.y * blockDim.x + threadIdx.x;

    // shared memory store operation
    tile[threadIdx.x][threadIdx.y] = idx;

    // wait for all threads to complete
    __syncthreads();

    // shared memory load operation
    out[idx] = tile[threadIdx.y][threadIdx.x];
}

__global__ void setRowReadCol(int *out)
{
    // static shared memory
    __shared__ int tile[BDIMY][BDIMX];

    // mapping from thread index to global memory index
    unsigned int idx = threadIdx.y * blockDim.x + threadIdx.x;

    // shared memory store operation
    tile[threadIdx.y][threadIdx.x] = idx;

    // wait for all threads to complete
    __syncthreads();

    // shared memory load operation
    out[idx] = tile[threadIdx.x][threadIdx.y];
}

__global__ void setRowReadColDyn(int *out)
{
    // dynamic shared memory
    extern  __shared__ int tile[];

    // mapping from thread index to global memory index
    unsigned int row_idx = threadIdx.y * blockDim.x + threadIdx.x;
    unsigned int col_idx = threadIdx.x * blockDim.y + threadIdx.y;

    // shared memory store operation
    tile[row_idx] = row_idx;

    // wait for all threads to complete
    __syncthreads();

    // shared memory load operation
    out[row_idx] = tile[col_idx];
}

__global__ void setColReadRowDyn(int *out)
{
    // dynamic shared memory
    extern  __shared__ int tile[];

    // mapping from thread index to global memory index
    unsigned int row_idx = threadIdx.y * blockDim.x + threadIdx.x;
    unsigned int col_idx = threadIdx.x * blockDim.y + threadIdx.y;

    // shared memory store operation
    tile[col_idx] = row_idx;

    // wait for all threads to complete
    __syncthreads();

    // shared memory load operation
    out[row_idx] = tile[row_idx];
}

__global__ void setRowReadRow(int *out){
    __shared__ int tile[BDIMX][BDIMY];

    unsigned int idx = threadIdx.y * blockDim.x + threadIdx.x;
    //共享内存存储操作
    tile[threadIdx.y][threadIdx.x] = idx;
    __syncthreads();
    //共享内存加载操作和全局内存存储操作
    out[idx] = tile[threadIdx.y][threadIdx.x];
}

int main{

};
//
// Created by 李亘杰 on 2021/3/1.
//

#include <string>
#include "sharedMemoryTest.cuh"

int main(){
    hipSharedMemConfig *pconfig;
    hipDeviceGetSharedMemConfig(pconfig);
    printf("%d\n",pconfig);

    hipSharedMemConfig config = hipSharedMemBankSizeFourByte;
    printf("config  %d\n",config);

    hipDeviceSetSharedMemConfig(config);
    hipDeviceGetSharedMemConfig(pconfig);

//    hipError_t errorCode = hipDeviceSetCacheConfig(hipFuncCachePreferEqual);
    printf("%d\n",pconfig);

}
#include "../common.h"
#include <stdio.h>
#include "../include/hip/hip_runtime.h"
#include "../include/"

#include <stdio.h>

#define MEGABYTE    (1024 * 1024)

int main(int argc, char **argv)
{
    // set up device
    int dev = 0;
    CHECK(hipSetDevice(dev));

    if (argc != 2) {
        printf("usage: %s <size-in-mbs>\n", argv[0]);
        return 1;
    }

    // memory size
    int n_mbs = atoi(argv[1]);
    unsigned int nbytes = n_mbs * MEGABYTE;

    // get device information
    hipDeviceProp_t deviceProp;
    CHECK(hipGetDeviceProperties(&deviceProp, dev));
    printf("%s starting at ", argv[0]);
    printf("device %d: %s nbyte %5.2fMB\n", dev,
           deviceProp.name, nbytes / (1024.0f * 1024.0f));

    // allocate the host memory
    double start = seconds();
    float *h_a = (float *)malloc(nbytes);
    double elapsed = seconds() - start;
    printf("Host memory allocation took %2.10f us\n", elapsed * 1000000.0);

    // allocate the device memory
    start = seconds();
    float *d_a;
    CHECK(hipMalloc((float **)&d_a, nbytes));
    elapsed = seconds() - start;
    printf("Allocate device memory allocation took %2.10f us\n", elapsed * 1000000.0);

    // initialize the host memory
    for(unsigned int i = 0; i < nbytes / sizeof(float); i++) h_a[i] = 0.5f;

    // transfer data from the host to the device
    start = seconds();
    CHECK(hipMemcpy(d_a, h_a, nbytes, hipMemcpyHostToDevice));
    elapsed = seconds() - start;
    printf("Transfer data from the host to the device took %2.10f us\n", elapsed * 1000000.0);

    // transfer data from the device to the host
    start = seconds();
    CHECK(hipMemcpy(h_a, d_a, nbytes, hipMemcpyDeviceToHost));
    elapsed = seconds() - start;
    printf("Transfer data from the device to the host took %2.10f us\n", elapsed * 1000000.0);

    // free memory
    start = seconds();
    CHECK(hipFree(d_a));
    elapsed = seconds() - start;
    printf("Free cuda memory took %2.10f us\n", elapsed * 1000000.0);


    start = seconds();
    free(h_a);
    elapsed = seconds() - start;
    printf("Host memory deallocation took %2.10f us\n", elapsed * 1000000.0);

    // reset device
    CHECK(hipDeviceReset());
    return EXIT_SUCCESS;
}

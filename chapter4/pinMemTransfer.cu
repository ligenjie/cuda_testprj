#include "../common.h"
#include <stdio.h>
#include "../include/hip/hip_runtime.h"
#include "../include/"

#include <stdio.h>

#define MEGABYTE    (1024 * 1024)

int main(int argc, char **argv)
{
    // set up device
    int dev = 0;
    CHECK(hipSetDevice(dev));

    if (argc != 2) {
        printf("usage: %s <size-in-mbs>\n", argv[0]);
        return 1;
    }

    // memory size
    int n_mbs = atoi(argv[1]);
    unsigned int nbytes = n_mbs * MEGABYTE;

    // get device information
    hipDeviceProp_t deviceProp;
    CHECK(hipGetDeviceProperties(&deviceProp, dev));

    if (!deviceProp.canMapHostMemory)
    {
        printf("Device %d does not support mapping CPU host memory!\n", dev);
        CHECK(hipDeviceReset());
        exit(EXIT_SUCCESS);
    }

    printf("%s starting at ", argv[0]);
    printf("device %d: %s nbyte %5.2fMB canMap %d\n", dev,
           deviceProp.name, nbytes / (1024.0f * 1024.0f),
           deviceProp.canMapHostMemory);

    // allocate pinned host memory
    float *h_a;
    double start = seconds();
    CHECK(hipHostMalloc ((float **)&h_a, nbytes));
    double elapsed = seconds() - start;
    printf("Host memory allocation took %2.10f us\n", elapsed * 1000000.0);

    // allocate device memory
    start = seconds();
    float *d_a;
    CHECK(hipMalloc((float **)&d_a, nbytes));
    elapsed = seconds() - start;
    printf("Allocate device memory allocation took %2.10f us\n", elapsed * 1000000.0);

    // initialize host memory
    memset(h_a, 0, nbytes);

    for (int i = 0; i < nbytes / sizeof(float); i++) h_a[i] = 100.10f;

    // transfer data from the host to the device
    // transfer data from the host to the device
    start = seconds();
    CHECK(hipMemcpy(d_a, h_a, nbytes, hipMemcpyHostToDevice));
    elapsed = seconds() - start;
    printf("Transfer data from the host to the device took %2.10f us\n", elapsed * 1000000.0);

    // transfer data from the device to the host
    start = seconds();
    CHECK(hipMemcpy(h_a, d_a, nbytes, hipMemcpyDeviceToHost));
    elapsed = seconds() - start;
    printf("Transfer data from the device to the host took %2.10f us\n", elapsed * 1000000.0);

    // free memory
    start = seconds();
    CHECK(hipFree(d_a));
    elapsed = seconds() - start;
    printf("Free cuda memory took %2.10f us\n", elapsed * 1000000.0);

    // reset device
    CHECK(hipDeviceReset());
    return EXIT_SUCCESS;
}
